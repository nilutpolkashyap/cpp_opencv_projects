
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <chrono>
#include <thread>

using namespace std;

constexpr int BLOCK_DIM = 16;

// determines if this node (pixel) is inside the circle
// result is stored in a [16*16] array
// thread 0 then computes the number of "in" nodes (value from 0 to 16*16)
__global__ void flagKernel(unsigned *block_counts) {
	bool __shared__ ins[BLOCK_DIM*BLOCK_DIM];
	// compute our coordinate in the global grid
	unsigned i = blockIdx.x*blockDim.x + threadIdx.x; // my i
	unsigned j = blockIdx.y*blockDim.y + threadIdx.y; // my j
	unsigned Ni = gridDim.x*blockDim.x;   // total number of nodes in x
	unsigned Nj = gridDim.y*blockDim.y;   // total number of nodex in y

	//get 1D index from i,j, u=j*ni+i
	unsigned u = threadIdx.y*blockDim.x + threadIdx.x;

	float x = i/(float)Ni;     // compute x in [0,1)
	float y = j/(float)Nj;     // y in [0,1)
	if (x*x+y*y<=1) ins[u] = true;  // check if in the circle
	else ins[u] = false;

	// wait for all threads in the block to finish
	__syncthreads();

	// let the first thread in the block add up "ins"
	if (u==0) {
		unsigned count = 0;
		for (int i=0;i<blockDim.x*blockDim.y;i++)
		  if (ins[u]) count++;

		// flattened index for the block, u=j*ni+i
		int block_u = blockIdx.y*gridDim.x+blockIdx.x;

		// store the sum in global memory
		block_counts[block_u] = count;
	}
}

// this kernel adds up block-level sums to the global sum
// this could be further optimized by splitting up the sum over threads
__global__ void addKernel(dim3 numBlocks, unsigned *block_counts, unsigned long *glob_count) {
	// compute total number of blocks
	unsigned N = numBlocks.x*numBlocks.y;
	unsigned long sum = 0;
	for (int i=0;i<N;i++)
		sum+=block_counts[i];

	// store result in global memory
	*glob_count = sum;
}


int main() {
  // grab starting time
  auto time_start = chrono::high_resolution_clock::now();

  // figure out how many samples I should process
  size_t N = BLOCK_DIM*1000;    // grid size

  // figure out our grid size
  dim3 threadsPerBlock(BLOCK_DIM, BLOCK_DIM);
  dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

  // allocate memory on the GPU
  unsigned *block_counts;
  hipMalloc((void**)&block_counts, numBlocks.x*numBlocks.y*sizeof(unsigned));

  unsigned long *N_in_gpu;  // GPU variable to hold the total N_in
  unsigned long N_in;		// CPU variable to hold this data
  hipMalloc((void**)&N_in_gpu, sizeof(N_in));

  // launch the kernel to flag nodes, each block has BLOCK_DIM*BLOCK_DIM threads
  flagKernel<<<numBlocks, threadsPerBlock>>>(block_counts);

  // launch kernel to add up per-block "in" counts
  addKernel<<<1, 1>>>(numBlocks, block_counts, N_in_gpu);

  // transfer N_in from the GPU to the CPU
  hipMemcpy(&N_in, N_in_gpu, sizeof(N_in), hipMemcpyDeviceToHost);

  auto time_now = chrono::high_resolution_clock::now();
  chrono::duration<double> time_delta = time_now-time_start;

  // compute pi and show the result on rank 0 (root) using the global data
  size_t N_tot = N*N;
  double pi = 4*N_in/(double)N_tot;
  cout<<"Using a "<<N<<"x"<<N<<" grid ("<<N_tot<<" samples), pi is "<<pi
	  <<" in "<<setprecision(3)<<time_delta.count()<<" seconds"<<endl;

  // be a good neighbor and free memory
  hipFree(block_counts);
  hipFree(N_in_gpu);

  return 0;
}


